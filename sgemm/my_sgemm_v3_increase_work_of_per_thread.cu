#include <assert.h>
#include <stdlib.h>
#include <cstdio>
#include <hip/hip_runtime.h>
#define A(i, j) a[(i) * n + (j)]
#define B(i, j) b[(i) * n + (j)]
void random_matrix(int m, int n, float *a)
{
    for (int i = 0; i < m; i++)
        for (int j = 0; j < n; j++)
#if 1
            A(i, j) = 2.0 * (float)drand48() - 1.0;
#else
            A(i, j) = (j - i) % 3;
#endif
}

float compare_matrices(int m, int n, float *a, float *b)
{
    int i, j;
    float max_diff = 0.0, diff;
    int printed = 0;
    for (i = 0; i < m; i++)
    {
        for (j = 0; j < n; j++)
        {
            diff = abs(A(i, j) - B(i, j));
            max_diff = (diff > max_diff ? diff : max_diff);
            if (0 == printed)
                if (max_diff > 0.5f || max_diff < -0.5f)
                {
                    printf("\n error: i %d j %d diff %f  got %f expect %f  ", i, j, max_diff, A(i, j), B(i, j));
                    printed = 1;
                }
        }
    }
    return max_diff;
}

void cpu_sgemm(float *A_ptr, float *B_ptr, float *C_ptr, const int M, const int N, const int K)
{
    for (int m = 0; m < M; m++)
    {
        for (int n = 0; n < N; n++)
        {
            float temp = 0.f;
            for (int k = 0; k < K; k++)
            {
                temp += A_ptr[m * K + k] * B_ptr[k * N + n];
            }
            C_ptr[m * N + n] = temp;
        }
    }
}

template <unsigned int BLOCK_SIZE, unsigned int STRIDE>
__global__ void cuda_sgemm(float *A_ptr, float *B_ptr, float *C_ptr, const int M, const int N, const int K)
{
    constexpr int STEP = BLOCK_SIZE * STRIDE;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    float *A_ptr_start = A_ptr + STEP * blockIdx.y * K;
    float *B_ptr_start = B_ptr + STEP * blockIdx.x;

    __shared__ float a_shared[STEP][STEP];
    __shared__ float b_shared[STEP][STEP];
    float temp[STRIDE][STRIDE] = {0.f};

    for (int s = 0; s < K; s += STEP)
    {
        // a_shared[threadIdx.y][threadIdx.x] = A_ptr_start[threadIdx.x + s + threadIdx.y * K];
        // b_shared[threadIdx.y][threadIdx.x] = B_ptr_start[(threadIdx.y + s) * N + threadIdx.x];
        for (int i = 0; i < STRIDE; i++)
        {
            for (int j = 0; j < STRIDE; j++)
            {
                a_shared[ty + i * BLOCK_SIZE][tx + j * BLOCK_SIZE] = A_ptr_start[(ty + BLOCK_SIZE * i) * K + tx + BLOCK_SIZE * j + s];
                b_shared[ty + i * BLOCK_SIZE][tx + j * BLOCK_SIZE] = B_ptr_start[(ty + BLOCK_SIZE * i + s) * N + tx + BLOCK_SIZE * j];
            }
        }
        __syncthreads();
        for (int i = 0; i < STRIDE; i++)
        {
            for (int j = 0; j < STRIDE; j++)
            {
                for (int k = 0; k < STEP; k++)
                {
                    temp[i][j] += a_shared[ty + i * BLOCK_SIZE][k] * b_shared[k][tx + j * BLOCK_SIZE];
                }
            }
        }
        __syncthreads();
    }
    float *C_ptr_start = C_ptr + N * blockIdx.y * STEP + blockIdx.x * STEP;
    for (int i = 0; i < STRIDE; i++)
    {
        for (int j = 0; j < STRIDE; j++)
        {
            C_ptr_start[N * (ty + i * BLOCK_SIZE) + tx + j * BLOCK_SIZE] = temp[i][j];
        }
    }
}

int main()
{
    int m = 1024;
    int n = 1024;
    int k = 1024;
    const size_t mem_size_A = m * k * sizeof(float);
    const size_t mem_size_B = m * k * sizeof(float);
    const size_t mem_size_C = m * k * sizeof(float);

    float *matrix_A_host = (float *)malloc(mem_size_A);
    float *matrix_B_host = (float *)malloc(mem_size_B);

    float *matrix_C_host_gpu_calc = (float *)malloc(mem_size_C);
    float *matrix_C_host_cpu_calc = (float *)malloc(mem_size_C);

    random_matrix(m, k, matrix_A_host);
    random_matrix(k, n, matrix_B_host);
    memset(matrix_C_host_cpu_calc, 0, mem_size_C);
    memset(matrix_C_host_gpu_calc, 0, mem_size_C);

    float *matrix_A_device, *matrix_B_device, *matrix_C_device;
    hipMalloc((void **)&matrix_A_device, mem_size_A);
    hipMalloc((void **)&matrix_B_device, mem_size_B);
    hipMalloc((void **)&matrix_C_device, mem_size_C);

    hipMemcpy(matrix_A_device, matrix_A_host, mem_size_A, hipMemcpyHostToDevice);
    hipMemcpy(matrix_B_device, matrix_B_host, mem_size_B, hipMemcpyHostToDevice);

    cpu_sgemm(matrix_A_host, matrix_B_host, matrix_C_host_cpu_calc, m, n, k);

    constexpr int BLOCK = 16;
    constexpr int STRIDE = 2;
    dim3 block(BLOCK, BLOCK);
    dim3 grid((m + BLOCK - 1) / BLOCK / STRIDE, (n + BLOCK - 1) / BLOCK / STRIDE);
    cuda_sgemm<BLOCK, STRIDE><<<grid, block>>>(matrix_A_device, matrix_B_device, matrix_C_device, m, n, k);

    hipMemcpy(matrix_C_host_gpu_calc, matrix_C_device, mem_size_C, hipMemcpyDeviceToHost);

    float diff = compare_matrices(m, n, matrix_C_host_gpu_calc, matrix_C_host_cpu_calc);
    if (diff > 0.5f || diff < -0.5f)
    {
        printf("diff too big ! \n");
        exit(-1);
    }
    else
    {
        printf("right\n");
    }

    free(matrix_A_host);
    free(matrix_B_host);
    free(matrix_C_host_cpu_calc);
    free(matrix_C_host_gpu_calc);

    hipFree(matrix_A_device);
    hipFree(matrix_B_device);
    hipFree(matrix_C_device);
    return 0;
}