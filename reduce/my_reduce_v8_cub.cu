#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include <time.h>
#include <sys/time.h>
#include <hipcub/hipcub.hpp>

bool check(float *out, float *res)
{
    if (abs(*out - *res) > 0.05)
        return false;
    return true;
}

int main()
{
    const int N = 32 * 1024 * 1024;
    float *a = (float *)malloc(N * sizeof(float));
    float *d_a;
    hipMalloc((void **)&d_a, N * sizeof(float));

    float *out = (float *)malloc(sizeof(float));
    float *d_out;
    hipMalloc((void **)&d_out, sizeof(float));
    float *res = (float *)malloc(sizeof(float));
    *res = 0.0f;

    for (int i = 0; i < N; i++)
    {
        a[i] = 2.0 * (float)drand48() - 1.0;
        *res += a[i];
    }

    hipMemcpy(d_a, a, N * sizeof(float), hipMemcpyHostToDevice);

    void *d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;

    hipcub::DeviceReduce::Reduce(
        d_temp_storage, temp_storage_bytes,
        d_a, d_out, N,
        hipcub::Sum(), 0.0f);

    hipMalloc(&d_temp_storage, temp_storage_bytes);

    hipcub::DeviceReduce::Reduce(
        d_temp_storage, temp_storage_bytes,
        d_a, d_out, N,
        hipcub::Sum(), 0.0f);

    hipMemcpy(out, d_out, sizeof(float), hipMemcpyDeviceToHost);

    if (check(out, res))
        printf("The answer is right.\n");
    else
    {
        printf("The answer is wrong.\n");
        printf("GPU result = %f, CPU result = %f\n", *out, *res);
    }

    hipFree(d_a);
    hipFree(d_out);
    hipFree(d_temp_storage);

    free(a);
    free(out);
    free(res);
}
