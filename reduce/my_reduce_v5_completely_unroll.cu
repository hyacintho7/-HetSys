#include <bits/stdc++.h>
#include <hip/hip_runtime.h>


#include <time.h>
#include <sys/time.h>

#define THREAD_PER_BLOCK 256

template <unsigned int blockSize>
__device__ void warpReduce(volatile float* cache, unsigned int tid){
    if (blockSize >= 64)cache[tid]+=cache[tid+32];
    if (blockSize >= 32)cache[tid]+=cache[tid+16];
    if (blockSize >= 16)cache[tid]+=cache[tid+8];
    if (blockSize >= 8)cache[tid]+=cache[tid+4];
    if (blockSize >= 4)cache[tid]+=cache[tid+2];
    if (blockSize >= 2)cache[tid]+=cache[tid+1];
}

template <unsigned int blockSize>
__global__ void reduce5(float *d_in,float *d_out){
    __shared__ float sdata[THREAD_PER_BLOCK];

    // each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
    sdata[tid] = d_in[i] + d_in[i + blockDim.x];
    __syncthreads();

    // do reduction in shared mem
//#pragma unroll
    if (blockSize >= 512) {
        if (tid < 256) { 
            sdata[tid] += sdata[tid + 256]; 
        } 
        __syncthreads(); 
    }
    if (blockSize >= 256) {
        if (tid < 128) { 
            sdata[tid] += sdata[tid + 128]; 
        } 
        __syncthreads(); 
    }
    if (blockSize >= 128) {
        if (tid < 64) { 
            sdata[tid] += sdata[tid + 64]; 
        } 
        __syncthreads(); 
    }
    if (tid < 32) warpReduce<blockSize>(sdata, tid);

    // write result for this block to global mem
    if (tid == 0) d_out[blockIdx.x] = sdata[0];
}

bool check(float *out,float *res,int n){
    for(int i=0;i<n;i++){
        if(out[i]!=res[i])
            return false;
    }
    return true;
}

int main(){
    const int N=32*1024*1024;
    float *a=(float *)malloc(N*sizeof(float));
    float *d_a;
    hipMalloc((void **)&d_a,N*sizeof(float));

    int NUM_PER_BLOCK = 2*THREAD_PER_BLOCK;
    int block_num = N / NUM_PER_BLOCK;
    float *out=(float *)malloc(block_num*sizeof(float));
    float *d_out;
    hipMalloc((void **)&d_out,block_num*sizeof(float));
    float *res=(float *)malloc(block_num*sizeof(float));

    for(int i=0;i<N;i++){
        a[i]=1;
    }

    for(int i=0;i<block_num;i++){
        float cur=0;
        for(int j=0;j<NUM_PER_BLOCK;j++){
            cur+=a[i * NUM_PER_BLOCK + j];
        }
        res[i]=cur;
    }

    hipMemcpy(d_a,a,N*sizeof(float),hipMemcpyHostToDevice);

    dim3 Grid( block_num, 1);
    dim3 Block( THREAD_PER_BLOCK, 1);

    reduce5<THREAD_PER_BLOCK><<<Grid,Block>>>(d_a,d_out);

    hipMemcpy(out,d_out,block_num*sizeof(float),hipMemcpyDeviceToHost);

    if(check(out,res,block_num))printf("the ans is right\n");
    else{
        printf("the ans is wrong\n");
        for(int i=0;i<block_num;i++){
            printf("%lf ",out[i]);
        }
        printf("\n");
    }

    hipFree(d_a);
    hipFree(d_out);
}